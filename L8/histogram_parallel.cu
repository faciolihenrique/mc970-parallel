// Henrique Noronha Facioli
// 157986


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>


#define COMMENT "Histogram_GPU"
#define RGB_COMPONENT_COLOR 255

typedef struct {
	unsigned char red, green, blue;
} PPMPixel;

typedef struct {
	int x, y;
	PPMPixel *data;
} PPMImage;

double rtclock()
{
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday (&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d",stat);
    return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}


static PPMImage *readPPM(const char *filename) {
	char buff[16];
	PPMImage *img;
	FILE *fp;
	int c, rgb_comp_color;
	fp = fopen(filename, "rb");
	if (!fp) {
		fprintf(stderr, "Unable to open file '%s'\n", filename);
		exit(1);
	}

	if (!fgets(buff, sizeof(buff), fp)) {
		perror(filename);
		exit(1);
	}

	if (buff[0] != 'P' || buff[1] != '6') {
		fprintf(stderr, "Invalid image format (must be 'P6')\n");
		exit(1);
	}

	img = (PPMImage *) malloc(sizeof(PPMImage));
	if (!img) {
		fprintf(stderr, "Unable to allocate memory\n");
		exit(1);
	}

	c = getc(fp);
	while (c == '#') {
		while (getc(fp) != '\n')
			;
		c = getc(fp);
	}

	ungetc(c, fp);
	if (fscanf(fp, "%d %d", &img->x, &img->y) != 2) {
		fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
		exit(1);
	}

	if (fscanf(fp, "%d", &rgb_comp_color) != 1) {
		fprintf(stderr, "Invalid rgb component (error loading '%s')\n",
				filename);
		exit(1);
	}

	if (rgb_comp_color != RGB_COMPONENT_COLOR) {
		fprintf(stderr, "'%s' does not have 8-bits components\n", filename);
		exit(1);
	}

	while (fgetc(fp) != '\n')
		;
	img->data = (PPMPixel*) malloc(img->x * img->y * sizeof(PPMPixel));

	if (!img) {
		fprintf(stderr, "Unable to allocate memory\n");
		exit(1);
	}

	if (fread(img->data, 3 * img->x, img->y, fp) != img->y) {
		fprintf(stderr, "Error loading image '%s'\n", filename);
		exit(1);
	}

	fclose(fp);
	return img;
}

__global__ void histogram_sum(PPMPixel *linear_image, float image_size, float *h) {
	int j, k, l;

	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i >= image_size)
		return;
	
	for (j = 0; j <= 3; j++) {
		for (k = 0; k <= 3; k++) {
			for (l = 0; l <= 3; l++) {
				if (linear_image[i].red   == j &&
					linear_image[i].green == k &&
					linear_image[i].blue  == l
				) {
					atomicAdd(&(h[(l+4*k+16*j)]), 1.0);
				}
			}
		}
	}
}

void Histogram(PPMImage *image, float *h) {
	int i;
	float n = image->y * image->x;
	PPMPixel *d_image;
	float *d_h;
	double t_start, t_end;

	for (i = 0; i < n; i++) {
		image->data[i].red = floor((image->data[i].red * 4) / 256);
		image->data[i].blue = floor((image->data[i].blue * 4) / 256);
		image->data[i].green = floor((image->data[i].green * 4) / 256);
	}

	t_start = rtclock();
	hipMalloc((void **) &d_image, sizeof(PPMPixel) * n);
	hipMalloc((void **) &d_h, sizeof(float) * 64);
	t_end = rtclock();
	fprintf(stdout, "criar_buff: %0.6lf\n", t_end - t_start);
	
	t_start = rtclock();
	hipMemset(d_h, 0.0, sizeof(float) * 64);
	hipMemcpy(d_image, image->data, sizeof(PPMPixel)*n, hipMemcpyHostToDevice);
	t_end = rtclock();
	fprintf(stdout, "enviar: %0.6lf\n", t_end - t_start);
	
	int n_blocks = ceil(n / 1024.0);
	dim3 gridDim(n_blocks);
	dim3 blockDim(1024);


	histogram_sum<<<gridDim, blockDim>>>(d_image, n, d_h);
	t_end = rtclock();
	fprintf(stdout, "kernel: %0.6lf\n", t_end - t_start);

	t_start = rtclock();
	hipMemcpy(h, d_h, sizeof(float) * 64, hipMemcpyDeviceToHost);
	t_end = rtclock();
	fprintf(stdout, "receber: %0.6lf\n", t_end - t_start);
}

int main(int argc, char *argv[]) {

	if( argc != 2 ) {
		printf("Too many or no one arguments supplied.\n");
	}

	double t_start, t_end;
	int i;
	char *filename = argv[1]; //Recebendo o arquivo!;
	
	//scanf("%s", filename);
	PPMImage *image = readPPM(filename);

	float *h = (float*)malloc(sizeof(float) * 64);

	//Inicializar h
	for(i=0; i < 64; i++) h[i] = 0.0;

	t_start = rtclock();
	Histogram(image, h);
	t_end = rtclock();

	for (i = 0; i < 64; i++){
		printf("%0.3f ", h[i]/(image->x * image->y));
	}
	printf("\n");
	fprintf(stdout, "%0.6lf\n", t_end - t_start);  
	free(h);
}
