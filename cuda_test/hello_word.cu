
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) { 
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x]; 
}

int main(void) {
	int a[4] = {2,2,2,2};
	int b[4] = {7,7,7,7};
	int c[4];
	int *d_a, *d_b, *d_c;
	int size = 4*sizeof(int);

	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<4,1>>>(d_a, d_b, d_c); 

	hipMemcpy(c, d_c, size,hipMemcpyDeviceToHost);

	printf("%d %d %d %d\n", c[0], c[1], c[2], c[3]);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0; 
}
